#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>

/* Include polybench common header. */
#include <polybench.h>

/* Include benchmark-specific header. */
/* Default data type is double, default size is 4000. */
#include "cholesky.h"

#define Nq N*N

#define BLOCK_SIZE 32

/* Array initialization. */
static void init_array(int n,
                       DATA_TYPE *p,
                       DATA_TYPE *A)
{
  int i, j;
  for (i = 0; i < n; i++)
  {
    p[i] = 0;   
    for (j = 0; j < n; j++)
      A[i*n + j] = 1.0 / n;

  }
}
              

/* Check the correctness of the two output. 
    If difference in output is found between A and A_d,
    it will be assert. */
static void check_correctness(int n, int nq,
                              DATA_TYPE *A_d,
                              DATA_TYPE *A)
{
  int i, j;
  for (i = 0; i < n; i++)
    for (j = 0; j < n; j++)
      assert(A[i*n + j] == A_d[i*n + j]);

  // If no assertion failures occurred, print a success message
  printf("Assertion passed: Each element in A is equal to the corresponding element in A_d.\n");
}

/* DCE code. Must scan the entire live-out data. */
static void print_dataset_matrix(int n,
                                DATA_TYPE *A)

{
  int i, j;

  for (i = 0; i < n; i++)
    for (j = 0; j < n; j++)
    {
      fprintf(stderr, DATA_PRINTF_MODIFIER, A[i*n + j]);
      if ((i * N + j) % 20 == 0)
        fprintf(stderr, "\n");
    }
}

/* DCE code. Must scan the entire live-out data. */
static void print_dataset_linear(int n, int nq,
                                 DATA_TYPE *A_d)
{
  int i, j;

  for (i = 0; i < n; i++)
    for (j = 0; j < n; j++)
    {
      fprintf(stderr, DATA_PRINTF_MODIFIER, A_d[i*n + j]);
      if ((i * N + j) % 20 == 0)
        fprintf(stderr, "\n");
    }
}

/* Main computational kernel. The whole function will be timed,
   including the call and return. */
static void kernel_cholesky(int n,
                            DATA_TYPE *p,
                            DATA_TYPE *A)
{
  int i, j, k;

  DATA_TYPE x;
  for (i = 0; i < _PB_N; ++i)
  {
    x = A[i*n + i];
    for (j = 0; j <= i - 1; ++j)
      x = x - A[i*n + j] * A[i*n + j];
    p[i] = 1.0 / sqrt(x);
    for (j = i + 1; j < _PB_N; ++j)
    {
      x = A[i*n + j];
      for (k = 0; k <= i - 1; ++k)
        x = x - A[j*n + k] * A[i*n + k];
      A[j*n + i] = x * p[i];
    }
  }
  //int i, j, k;
//
  //DATA_TYPE x;
  //  for (i = 0; i < _PB_N; ++i) {
  //      p[i] = 1 / sqrt(A[i*n + i] - p[i]);
//
  //      #pragma omp parallel for private(j, k, x)
  //      for (j = i + 1; j < _PB_N; j++) {
  //          x = A[i*n + j];
  //          
  //          #pragma omp simd reduction(-:x)
  //          for (k = 0; k <= i - 1; ++k)
  //              x = x - A[j*n + k] * A[i*n + k];
//
  //          A[j*n + i] = x * p[i];
  //          p[j] += A[j*n + i] * A[j*n + i]; 
  //      }
  //  }
}
__global__ void device_cholesky_1(int n,
                                int i,
                                DATA_TYPE *p,
                                DATA_TYPE *A) 
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (tid == 0)
    p[i] = A[i * n + i];
  
  __syncthreads();

  DATA_TYPE tmp = 0;
  for (int j = 0; j < i; j += BLOCK_SIZE) {
    int index = j + tid;
    if (index < i) 
      tmp -= A[i * n + index] * A[i * n + index];
  }

  atomicAdd(&p[i], -tmp);

  __syncthreads();

  if (tid == 0) 
    p[i] = 1 / sqrt(p[i]);
}

__global__ void device_cholesky_2(int n,
                                int i,
                                DATA_TYPE *p,
                                DATA_TYPE *A) 
{
  int j = blockIdx.x * blockDim.x + threadIdx.x + i + 1;
  if (j >= n)
    return;

  DATA_TYPE tmp = A[i*n + j];
  for (int k = 0; k < i; k++)
    tmp -= A[i*n + k] * A[j*n + k];
  
  A[j*n + i] = p[i] * tmp;
}

int main(int argc, char **argv)
{
  /* Retrieve problem size. */
  int n = N;
  int nq = N*N;

  /* Variable declaration/allocation. */
  DATA_TYPE *p, *A, *A_d;

  /* Allocate pinned memory on the host. */
  hipHostAlloc((void**)&p, N * sizeof(DATA_TYPE), hipHostMallocDefault);
  hipHostAlloc((void**)&A, N * N * sizeof(DATA_TYPE), hipHostMallocDefault);
  hipHostAlloc((void**)&A_d, N * N * sizeof(DATA_TYPE), hipHostMallocDefault);

  /* Allocate device memory */
  DATA_TYPE *d_p, *d_A;
  hipMalloc((void**)&d_p, N * sizeof(DATA_TYPE));
  hipMalloc((void**)&d_A, N * N * sizeof(DATA_TYPE));

  /* Initialize array(s). */
  init_array(n, p, A);

  hipMemcpy(A_d, A, N * N * sizeof(DATA_TYPE), hipMemcpyHostToHost);

  /* Start timer. */
  polybench_start_instruments;

  /* Run kernel. */
  kernel_cholesky(n, p, A);

  /* Stop and print timer. */
  polybench_stop_instruments;
  polybench_print_instruments;

  hipMemset(&p, 0, N * sizeof(DATA_TYPE));
  /* Run GPU kernel. */

  polybench_start_instruments;
  /* Copy data from pinned host memory to device memory. */
  hipMemcpy(d_p, p, N * sizeof(DATA_TYPE), hipMemcpyHostToDevice);
  hipMemcpy(d_A, A_d, Nq * sizeof(DATA_TYPE), hipMemcpyHostToDevice);
  
  for (int i = 0; i < N; i++) {
    device_cholesky_1<<<1, BLOCK_SIZE>>>(n, i, d_p, d_A);

    if (i < n - 1) {
      int numBlocks = (N - i + BLOCK_SIZE) / BLOCK_SIZE;
      device_cholesky_2<<<numBlocks, BLOCK_SIZE>>>(n, i, d_p, d_A);
    }
  }
  /* Copy results from device memory to pinned host memory. */
  hipMemcpy(p, d_p, N * sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
  hipMemcpy(A_d, d_A, Nq * sizeof(DATA_TYPE), hipMemcpyDeviceToHost);

  polybench_stop_instruments;
  polybench_print_instruments;

  //print_dataset_matrix(n, A_d);
  //    fprintf(stderr, "\n----------------------\n");
  //print_dataset_matrix(n, A);
  /* Check the correctness of the CPU and GPU/Device implementation. */
  check_correctness(n, nq, A_d, A);

  /* Prevent dead-code elimination. All live-out data must be printed
     by the function call in argument. */
  // polybench_prevent_dce(print_dataset_matrix(n, POLYBENCH_ARRAY(A)));
  // polybench_prevent_dce(print_dataset_linear(n, nq, POLYBENCH_ARRAY(A_d)));

  /* Free device memory. */
  hipFree(d_p);
  hipFree(d_A);

  /* Be clean. */
  hipHostFree(p);
  hipHostFree(A);
  hipHostFree(A_d);

  return 0;
}
